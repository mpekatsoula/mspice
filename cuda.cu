#include "hip/hip_runtime.h"
#include <cusp/coo_matrix.h>
#include <cusp/print.h>
#include <cusp/array1d.h>
#include <cusp/krylov/cg.h>
#include <cusp/precond/aggregation/smoothed_aggregation.h>
#include <cusp/krylov/detail/bicg.inl>


extern "C" {
#include "hms/error_header.h"
#include "hms/csparse.h"
#include "hms/solution_functions.h"
}

clock_t start, end;

void start_timer() {

  start = clock();
}

double end_timer() {

  return ((double)clock() - start ) / CLOCKS_PER_SEC;
}
// 44943
// 127565
// 852536
// 954542
// 1618394
// 2506730

int it1;
cusp::coo_matrix<int,double,cusp::host_memory> cuMNA( 2506730,2506730, 7504532); //147315*2

extern "C"
void cuda_init () {

  it1 = 0;
  
}

extern "C"
void cuda_implementation () {
printf("%d\n",it1);
    cusp::array1d<double, cusp::device_memory> b (MNA_matrix_size );
    cusp::array1d<double, cusp::device_memory> x (MNA_matrix_size );

    int i;
    for ( i = 0; i < MNA_matrix_size; i ++ )
      b[i] = b_sparse_vector[i];
  
    cusp::csr_matrix<int,double,cusp::device_memory> A = cuMNA;
    cusp::default_monitor<double> monitor(b, 36520, 1e-6);
    cusp::precond::aggregation::smoothed_aggregation<int, double, cusp::device_memory> M(A);
    start_timer();
    cusp::krylov::bicg(A,A, x,b, monitor, M, M);
   // cusp::krylov::cg(A, x,b, monitor,M);
    printf("time %g\n",end_timer());
   // cusp::print(x);

}

extern "C"
void add_cuda_element ( int idx1, int idx2, double val ) {

 // if ( temp[idx1][idx2] != -1 ) {

   // cuMNA.values[temp[idx1][idx2]] += val;

 // } 
 // else {
//
 //   temp[idx1][idx2] = it1;
    int i = 0;
    for ( i = 0; i < it1; i ++ ) {
      if ( cuMNA.row_indices[i] == idx1 && cuMNA.row_indices[i] == idx2  ) {
   //     cuMNA.row_indices[it1] = idx1;
    //    cuMNA.column_indices[it1] = idx2;
        cuMNA.values[i] += val;
        return;
      }
    }
        cuMNA.row_indices[it1] = idx1;
        cuMNA.column_indices[it1] = idx2;
        cuMNA.values[it1] = val;    
        
    it1++;
    if ( !(it1 %50000) )
      printf("it %d\n", it1);
 // }
  
}

extern "C"
void add_cuda_b ( int idx, double val ) {
  
  //b[idx] = val;

}
